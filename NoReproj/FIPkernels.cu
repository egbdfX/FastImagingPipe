#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>
#include <hipfft/hipfft.h>

#define M_PI 3.14159265358979323846

/* The gridding kernels are developed based on SKA SDP (https://gitlab.com/ska-telescope/sdp/ska-sdp-func). */

__constant__ float quadrature_nodes[14] = {
	0.9964425,0.98130317,0.95425928,0.91563303,0.86589252,
	0.80564137,0.73561088,0.65665109,0.56972047,0.47587422,
	0.37625152,0.27206163,0.16456928,0.05507929
};
__constant__ float quadrature_weights[14] = {
	0.00912428,0.02113211,0.03290143,0.04427293,0.05510735,
	0.06527292,0.07464621,0.08311342,0.09057174,0.09693066,
	0.10211297,0.10605577,0.10871119,0.11004701
};
__constant__ float quadrature_kernel[14] = {
	7.71381676e-07,4.06901586e-06,2.09164257e-05,1.01923695e-04,
	4.61199576e-04,1.90183990e-03,7.02391280e-03,2.28652529e-02,
	6.46725327e-02,1.56933676e-01,3.23208771e-01,5.60024174e-01,
	8.10934691e-01,9.76937533e-01
};

float computeCeil(float num) {
    if (num<0) {
		return -floorf(-num);
	} else {
		return ceilf(num);
	}
}

float computeFloor(float num) {
    if (num<0) {
		return -ceilf(-num);
	} else {
		return floorf(num);
	}
}

__device__ float ceil_device(float num) {
    if (num<0) {
		return -floorf(-num);
	} else {
		return ceilf(num);
	}
}

__device__ float floor_device(float num) {
    if (num<0) {
		return -ceilf(-num);
	} else {
		return floorf(num);
	}
}

__device__ float exp_semicircle(const float beta, float x){
    const float xx = x*x;
    
    return ((xx > float(1.0)) ? float(0.0) : exp(beta*(sqrt(float(1.0) - xx) - float(1.0))));
}

__global__ void convolveKernel(float *conv_corr_kernel, size_t image_size, size_t grid_size, float conv_corr_norm_factor) {
    const int support = 8;
    size_t t1_t2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (t1_t2 < image_size / 2 + 1) {
        float t1_t2_norm = static_cast<float>(t1_t2) / grid_size;
        float correction = 0.0;
        float angle;
        for (int i = 0; i < 14; ++i) {
            angle = M_PI * t1_t2_norm * support * quadrature_nodes[i];
            correction += quadrature_kernel[i] * quadrature_weights[i] * cosf(angle);
        }
        conv_corr_kernel[t1_t2] = correction * support / conv_corr_norm_factor;
    }
}

__global__ void computeVisWeighted(float *Vis_real, float *Vis_imag, size_t num_baselines, float inten_scale) {
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_baselines) {
		Vis_real[idx] = Vis_real[idx]/inten_scale;
		Vis_imag[idx] = Vis_imag[idx]/inten_scale;
	}
}

__global__ void gridding(float* B_in, float* w_grid_stack_real, float* w_grid_stack_imag, float* Vis_real, float* Vis_imag, float freq_hz, float uv_scale, size_t grid_size, size_t num_baselines) {
	float inv_wavelength = freq_hz / 299792458;
	const int support = 8;
	int half_support = support / 2;
	float inv_half_support = 1 / static_cast<float>(half_support);
    long int grid_min_uv = -static_cast<long int>(grid_size) / 2;
    long int grid_max_uv = (static_cast<long int>(grid_size) - 1) / 2;
    long int origin_offset_uv = static_cast<long int>(grid_size) / 2;
    const int KERNEL_SUPPORT_BOUND = 16;
    const float beta = 15.3704324328;
    float kernel_value;
	
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < num_baselines) {
		float pos_u = B_in[idx*2+0] * inv_wavelength * uv_scale;
		float pos_v = B_in[idx*2+1] * inv_wavelength * uv_scale;
		long int grid_u_min = max(static_cast<long int>(ceil_device(pos_u - half_support)), grid_min_uv);
        long int grid_u_max = min(static_cast<long int>(floor_device(pos_u + half_support)), grid_max_uv);
        long int grid_v_min = max(static_cast<long int>(ceil_device(pos_v - half_support)), grid_min_uv);
        long int grid_v_max = min(static_cast<long int>(floor_device(pos_v + half_support)), grid_max_uv);
        if (grid_u_min > grid_u_max || grid_v_min > grid_v_max) {
            return;
        }
		float kernel_u[KERNEL_SUPPORT_BOUND], kernel_v[KERNEL_SUPPORT_BOUND];
		for (long int grid_u = grid_u_min; grid_u <= grid_u_max; grid_u++)
		{
			kernel_u[grid_u - grid_u_min] = exp_semicircle(beta,(static_cast<float>(grid_u) - pos_u) * inv_half_support);
		}
		for (long int grid_v = grid_v_min; grid_v <= grid_v_max; grid_v++)
		{
			kernel_v[grid_v - grid_v_min] = exp_semicircle(beta,(static_cast<float>(grid_v) - pos_v) * inv_half_support);
		}
		
        for (long int grid_u = grid_u_min; grid_u <= grid_u_max; grid_u++)
        {
            for (long int grid_v = grid_v_min; grid_v <= grid_v_max; grid_v++)
            {
                kernel_value = kernel_u[grid_u - grid_u_min] * kernel_v[grid_v - grid_v_min];
                if (((grid_u + grid_v) & 1) != 0) {
					kernel_value = -kernel_value;
				}
                const long int grid_offset_uvw = (grid_u + origin_offset_uv) * static_cast<long int>(grid_size) + (grid_v + origin_offset_uv);
                        
                atomicAdd(&w_grid_stack_real[grid_offset_uvw],Vis_real[idx] * kernel_value);
                atomicAdd(&w_grid_stack_imag[grid_offset_uvw],Vis_imag[idx] * kernel_value);
                }
		}
	}
}

__global__ void combineToComplex(float* w_real, float* w_imag, hipfftComplex* complex_data, size_t grid_size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t size = grid_size * grid_size;
    if (idx < size) {
        complex_data[idx].x = w_real[idx];
        complex_data[idx].y = w_imag[idx];
    }
}

__global__ void ifftShift(hipfftComplex* data, hipfftComplex* data_shifted, size_t NX, size_t NY) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < NX && idy < NY) {
        size_t new_x = (idx + NX / 2) % NX;
        size_t new_y = (idy + NY / 2) % NY;
        size_t old_id = idy * NX + idx;
        size_t new_id = new_y * NX + new_x;
        
        data_shifted[new_id] = data[old_id];
    }
}

__global__ void accumulation(float* dirty_pre, hipfftComplex* w_grid_stack_shifted, size_t image_size, size_t grid_size) {
	size_t half_image_size = image_size / 2;
	size_t grid_index_offset_image_centre = grid_size*grid_size/2 + grid_size/2;
	size_t image_index_offset_image_centre = half_image_size*image_size + half_image_size;
	long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < image_size && idy < image_size) { 
		idx = idx - half_image_size;
		idy = idy - half_image_size;
		float pixel_sum = w_grid_stack_shifted[grid_index_offset_image_centre + idy*grid_size + idx].x;
        if (((abs(idx)+abs(idy)) & 1) != 0) {
			pixel_sum = - pixel_sum;
		}
		dirty_pre[image_index_offset_image_centre + idy*image_size + idx] += pixel_sum;
    }
}

__global__ void scaling(float* dirty_pre, float* conv_corr_kernel, size_t image_size, float conv_corr_norm_factor) {
	size_t half_image_size = image_size / 2;
	size_t image_index_offset_image_centre = half_image_size*image_size + half_image_size;
	long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long int idy = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (idx < image_size && idy < image_size) { 
		idx = idx - half_image_size;
		idy = idy - half_image_size;
        
        dirty_pre[image_index_offset_image_centre + idy * image_size + idx] *= 1/(conv_corr_kernel[abs(idx)]*conv_corr_kernel[abs(idy)]*conv_corr_norm_factor*conv_corr_norm_factor);
		dirty_pre[image_index_offset_image_centre + idy * image_size + idx] = abs(dirty_pre[image_index_offset_image_centre + idy * image_size + idx]);
	}
}

__global__ void coordschange(float* output_index, float* V_in, size_t image_size) {
	long int half_image_size = (long int)image_size / 2;
	
	long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < image_size && idy < image_size) {
        output_index[(idx*image_size+idy)*2+0] = (-V_in[0*3+0]*(idx - half_image_size)+V_in[1*3+0]*(idy - half_image_size))/abs(V_in[2*3+2]) + half_image_size;
        output_index[(idx*image_size+idy)*2+1] = (-V_in[0*3+1]*(idx - half_image_size)+V_in[1*3+1]*(idy - half_image_size))/abs(V_in[2*3+2]) + half_image_size;
	}
}

__global__ void finalinterp(float* output_index, float* dirty_pre, float* dirty, size_t image_size) {
	long int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long int idy = blockIdx.y * blockDim.y + threadIdx.y;
    size_t half_image_size = image_size / 2;
    size_t image_index_offset_image_centre = half_image_size*image_size + half_image_size;
    
    if (idx < image_size && idy < image_size) {
		float LL = output_index[(idx*image_size+idy)*2+0] - half_image_size;
		float MM = output_index[(idx*image_size+idy)*2+1] - half_image_size;
		
		idx = idx - half_image_size;
		idy = idy - half_image_size;
		
		if (abs(LL) < half_image_size-1 && abs(MM)<half_image_size-1) {
			atomicAdd(
				&dirty[
					static_cast<long int>(
						image_index_offset_image_centre+floor_device(MM)*image_size+floor_device(LL)
					)
				],
				(1-LL+floor_device(LL))*(1-MM+floor_device(MM))*
					dirty_pre[
						static_cast<long int>(
							image_index_offset_image_centre+idy*image_size+idx
						)
					]
			);
            atomicAdd(
				&dirty[
					static_cast<long int>(
						image_index_offset_image_centre+ceil_device(MM)*image_size+floor_device(LL)
					)
				],
				(1-LL+floor_device(LL))*(MM-floor_device(MM))*
					dirty_pre[
						static_cast<long int>(
							image_index_offset_image_centre+idy*image_size+idx
						)
					]
			);
            atomicAdd(
				&dirty[
					static_cast<long int>(
						image_index_offset_image_centre+floor_device(MM)*image_size+ceil_device(LL)
					)
				],
				(LL-floor_device(LL))*(1-MM+floor_device(MM))*
					dirty_pre[
						static_cast<long int>(
							image_index_offset_image_centre+idy*image_size+idx
						)
					]
			);
            atomicAdd(
				&dirty[
					static_cast<long int>(
						image_index_offset_image_centre+ceil_device(MM)*image_size+ceil_device(LL)
					)
				],
				(LL-floor_device(LL))*(MM-floor_device(MM))*
					dirty_pre[
						static_cast<long int>(
							image_index_offset_image_centre+idy*image_size+idx
						)
					]
			);
		}
	}
}

int FIpipe(float* Visreal, float* Visimag, float* Bin, float* Vin, float* dirty_image, size_t num_baselines, size_t image_size, float freq_hz, float cell_size){
	float* Vis_real;
	float* Vis_imag;
	float* B_in;
	float* V_in;
	float* dirty;
	float* dirty_pre;
	float* conv_corr_kernel;
	float* w_grid_stack_real;
	float* w_grid_stack_imag;
	float* pixel_ind;
	hipError_t cudaStatus;
	hipfftComplex* w_grid_stack;
	hipfftComplex* w_grid_stack_shifted;
	float* output_index;
	hipError_t hipError_t;
	double *h_output_index = new double[image_size * image_size * 2];
    for (size_t i = 0; i < image_size * image_size * 2; i++) {
        h_output_index[i] = 0.0;
    }
	
	hipEvent_t start, stop, eventstream;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&eventstream);
	
	size_t grid_size = computeCeil(1.5*static_cast<float>(image_size));
	float uv_scale = cell_size*grid_size;
	float conv_corr_norm_factor = 2.4937047051153827;
	
	hipMalloc((void**)&Vis_real, num_baselines * 1 * sizeof(float));
	hipMalloc((void**)&Vis_imag, num_baselines * 1 * sizeof(float));
	hipMalloc((void**)&B_in, num_baselines * 2 * sizeof(float));
	hipMalloc((void**)&V_in, 3 * 3 * sizeof(float));
	hipMalloc((void**)&dirty, image_size * image_size * sizeof(float));
	hipMalloc((void**)&dirty_pre, image_size * image_size * sizeof(float));
	hipMalloc((void**)&conv_corr_kernel, (image_size/2+1)*sizeof(float));
	hipMalloc((void**)&w_grid_stack_real, grid_size * grid_size * sizeof(float));
	hipMalloc((void**)&w_grid_stack_imag, grid_size * grid_size * sizeof(float));
	hipMalloc((void**)&w_grid_stack, grid_size * grid_size * sizeof(hipfftComplex));
	hipMalloc((void**)&w_grid_stack_shifted, grid_size * grid_size * sizeof(hipfftComplex));
	hipMalloc((void**)&output_index, image_size * image_size * 2 * sizeof(float));
	hipMalloc((void**)&pixel_ind, image_size * image_size * 2 * sizeof(float));
	
	hipMemcpy(Vis_real, Visreal, num_baselines * 1 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Vis_imag, Visimag, num_baselines * 1 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_in, Bin, num_baselines * 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(V_in, Vin, 3 * 3 * sizeof(float), hipMemcpyHostToDevice); // cross term included
	hipMemcpy(dirty, dirty_image, image_size * image_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(dirty_pre, 0.0, image_size * image_size * sizeof(float));
	hipMemset(conv_corr_kernel, 0.0, (image_size/2+1) * sizeof(float));
	hipMemset(w_grid_stack_real, 0.0, grid_size * grid_size * sizeof(float));
	hipMemset(w_grid_stack_imag, 0.0, grid_size * grid_size * sizeof(float));
	hipMemset(output_index, 0.0, image_size * image_size * 2 * sizeof(float));

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error 1 : %s\n", hipGetErrorString(cudaStatus));
	}
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 1 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 1.\n");
	}
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	
	hipEventRecord(start);
	/* ****************************************************** */
	size_t num_threads = 1024;
	size_t num_blocks = computeCeil(static_cast<float>(image_size/2+1)/num_threads);
	convolveKernel<<<num_blocks,num_threads,0,stream2>>>(conv_corr_kernel, image_size, grid_size, conv_corr_norm_factor);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 2 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 2.\n");
	}
	
	float inten_scale = std::abs(Vin[0*3+0]*Vin[1*3+1]-Vin[0*3+1]*Vin[1*3+0]);
	
	/* ****************************************************** */
	num_threads = 1024;
	num_blocks = computeCeil(static_cast<float>(num_baselines)/num_threads);
	computeVisWeighted<<<num_blocks,num_threads,0,stream1>>>(Vis_real,Vis_imag,num_baselines,inten_scale);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 3 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 3.\n");
	}
	
	/* ****************************************************** */
	num_threads = 1024;
	num_blocks = computeCeil(static_cast<float>(num_baselines)/num_threads);
	gridding<<<num_blocks,num_threads,0,stream1>>>(B_in, w_grid_stack_real, w_grid_stack_imag, Vis_real, Vis_imag, freq_hz, uv_scale, grid_size, num_baselines);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 4 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 4.\n");
	}
	
	/* ****************************************************** */
	num_threads = 1024;
	num_blocks = computeCeil(static_cast<float>(grid_size * grid_size)/num_threads);
	combineToComplex<<<num_blocks,num_threads,0,stream1>>>(w_grid_stack_real, w_grid_stack_imag, w_grid_stack, grid_size);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 5 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 5.\n");
	}
	
	/* ****************************************************** */
	num_threads = 32;
	dim3 numThreads(num_threads, num_threads);
	dim3 numBlocks(computeCeil(static_cast<float>(grid_size)/num_threads), computeCeil(static_cast<float>(grid_size)/num_threads));
    ifftShift<<<numBlocks,numThreads,0,stream1>>>(w_grid_stack, w_grid_stack_shifted, grid_size, grid_size);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 6 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 6.\n");
	}
    
    /* ****************************************************** */
    hipfftHandle plan;
    hipfftCreate(&plan);
	hipfftSetStream(plan, stream1);
    hipfftPlan2d(&plan, grid_size, grid_size, HIPFFT_C2C);

    hipfftExecC2C(plan, w_grid_stack_shifted, w_grid_stack_shifted, HIPFFT_BACKWARD);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 7 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 7.\n");
	}
	
	/* ****************************************************** */
	numThreads.x = num_threads;
	numThreads.y = num_threads;
    numBlocks.x = computeCeil(static_cast<float>(image_size)/num_threads);
    numBlocks.y = computeCeil(static_cast<float>(image_size)/num_threads);
    accumulation<<<numBlocks,numThreads,0,stream1>>>(dirty_pre, w_grid_stack_shifted, image_size, grid_size);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 8 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 8.\n");
	}
    
    /* ****************************************************** */
    numThreads.x = num_threads;
    numThreads.y = num_threads;
    numBlocks.x = computeCeil(static_cast<float>(image_size)/num_threads);
    numBlocks.y = computeCeil(static_cast<float>(image_size)/num_threads);
    scaling<<<numBlocks,numThreads,0,stream1>>>(dirty_pre, conv_corr_kernel, image_size, conv_corr_norm_factor);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 9 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 9.\n");
	}
	
	/* ****************************************************** */
	numThreads.x = num_threads;
	numThreads.y = num_threads;
    numBlocks.x = computeCeil(static_cast<float>(image_size)/num_threads);
    numBlocks.y = computeCeil(static_cast<float>(image_size)/num_threads);
	coordschange<<<numBlocks,numThreads,0,stream2>>>(output_index, V_in, image_size);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 10 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 10.\n");
	}
	hipEventRecord(eventstream,stream2);
	
	hipStreamWaitEvent(stream1,eventstream,0);
	
	/* ****************************************************** */
	numThreads.x = num_threads;
	numThreads.y = num_threads;
    numBlocks.x = computeCeil(static_cast<float>(image_size)/num_threads);
    numBlocks.y = computeCeil(static_cast<float>(image_size)/num_threads);
    finalinterp<<<numBlocks,numThreads,0,stream1>>>(output_index, dirty_pre, dirty, image_size);
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 14 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 14.\n");
	}
	
	hipStreamSynchronize(stream1);
	
	hipEventDestroy(eventstream);
	hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);
    
    hipMemcpy(dirty_image, dirty, image_size * image_size * sizeof(float), hipMemcpyDeviceToHost);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error 15 : %s\n", hipGetErrorString(cudaStatus));
	}
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess){
		printf("ERROR! GPU Kernel 15 error.\n");
		printf("CUDA error code: %d; string: %s;\n", (int) hipError_t, hipGetErrorString(hipError_t));
	}
	else {
		printf("No CUDA error 15.\n");
	}
	
	hipFree(Vis_real);
	hipFree(Vis_imag);
	hipFree(B_in);
	hipFree(V_in);
	hipFree(dirty);
	hipFree(dirty_pre);
	hipFree(conv_corr_kernel);
	hipFree(w_grid_stack_real);
	hipFree(w_grid_stack_imag);
	hipFree(w_grid_stack);
	hipFree(w_grid_stack_shifted);
	hipFree(output_index);
	
	return 0;
}
	
